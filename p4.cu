#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOQUE 5

__global__ void info (int *vector, int *resultado, int desplazamiento, int tamano) {
	int gId = threadIdx.x + blockDim.x * blockIdx.x;
	if (gId<tamano)
		resultado[gId] = vector[((-desplazamiento/tamano + 1)*tamano+gId+desplazamiento)%tamano];
}

int main(int argc, char** argv) {
	printf("Introduce el tamaño del vector: ");
	int tamano;
	scanf("%d",&tamano);
	printf("\nIntroduce el desplazamiento: ");
	int desplazamiento;
	scanf("%d",&desplazamiento);
	printf("\n\n tamano: %d\n", tamano);

	int *vector, *resultado;
	int *dev_vector, *dev_resultado; // reserva en el host
	
	// declaracion de eventos
	hipEvent_t start;
	hipEvent_t stop;
	// creacion de eventos
	hipEventCreate(&start);
	hipEventCreate(&stop);

	vector = (int *)malloc(tamano*sizeof(int));
	resultado = (int *)malloc(tamano*sizeof(int));

	// reserva en el device
	hipMalloc( (void**)&dev_vector, tamano*sizeof(int));
	hipMalloc( (void**)&dev_resultado, tamano*sizeof(int));
	for(int i=0;i<tamano;i++){
		vector[i]=2*i+1;
	}
	hipMemcpy(dev_vector, vector, tamano*sizeof(int), hipMemcpyHostToDevice);

	hipEventRecord(start,0);
	info<<<(tamano + BLOQUE - 1)/BLOQUE,BLOQUE>>>(dev_vector, dev_resultado, desplazamiento, tamano);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,start,stop);

	hipMemcpy(resultado, dev_resultado, tamano*sizeof(int), hipMemcpyDeviceToHost);

	printf("Vector de entrada\n");
	for(int i=0;i<tamano;i++){
		printf("%2d ", vector[i]);
	}
  
	printf("\nVector de salida\n");
	for(int i=0;i<tamano;i++){
		printf("%2d ", resultado[i]);
	}

	printf("\n> Tiempo de ejecucion: %f ms\n",elapsedTime);

	// liberacion de recursos
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree( dev_resultado );
	hipFree( dev_vector );

	// salida
	printf("\npulsa INTRO para finalizar..."); fflush(stdin);
	char tecla = getchar();
	return 0;
}
